#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "pyutils/pyutils.cuh"
using namespace kittens;

using my_layout = gl<float, -1, -1, -1, 64, st_fl<64,64>>; // An example layout that also instantiates a TMA descriptor on Hopper.
struct globals {
    my_layout in, out;
    __host__ dim3 grid()  { return dim3(in.batch(), in.depth(), in.rows()); }
    __host__ dim3 block() { return dim3(in.cols()); }
};
__global__ void copy_kernel(const __grid_constant__ globals g) {
    if(threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0) printf("Hello, from inside the kernel!\n");
    g.out[{blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x}] = g.in[{blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x}];
}
void run_copy_kernel(globals g) {
    printf("I am calling the kernel from the host.\n");
    copy_kernel<<<g.grid(), g.block()>>>(g);
}

PYBIND11_MODULE(example_bind, m) {
    m.doc() = "example_bind python module";
    py::bind_kernel<copy_kernel>(m, "copy_kernel", &globals::in, &globals::out);
    py::bind_function<run_copy_kernel>(m, "wrapped_copy_kernel", &globals::in, &globals::out);
}
