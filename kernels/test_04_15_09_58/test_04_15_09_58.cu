#include "hip/hip_runtime.h"
#include <iostream>
#include "kittens.cuh"
#include "pyutils/pyutils.cuh"

#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError( const char *file, const int line ) {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
}

using namespace kittens;

constexpr int PIPE_STAGES = 2;
constexpr int TILE_SIZE_N = 16;
constexpr int TILE_SIZE_M = 32;
constexpr int TILE_SIZE_D = 128;
// constexpr int QO_SEQ = TILE_SIZE_N;
// constexpr int KV_BLOCKS = 32;
// constexpr int KV_SEQ = TILE_SIZE_N * KV_BLOCKS;

template<typename T=bf16, typename L=row_l> using kv_tile = rt<T, TILE_SIZE_N, TILE_SIZE_D, L>;
template<typename T=bf16, typename L=row_l> using qo_tile = rt<T, TILE_SIZE_M, TILE_SIZE_D, L>;
template<typename T=float> using attn_tile = rt<T, TILE_SIZE_M, TILE_SIZE_N>;
using shared_kv_tile = st_bf<TILE_SIZE_N, TILE_SIZE_D>;
using shared_qo_tile = st_bf<TILE_SIZE_M, TILE_SIZE_D>;
using global_qkvo_layout = gl<bf16, -1, -1, -1, TILE_SIZE_D>; // batch, depth, row, col
struct globals {
    global_qkvo_layout Qg, Kg, Vg, Og;
};

__launch_bounds__(WARP_THREADS, 1)
__global__ void attend_ker(const __grid_constant__ globals g) {
    const int batch = blockIdx.y, head = blockIdx.x;

    extern __shared__ alignment_dummy __shm[];
    shared_allocator al((int*)&__shm[0]);

    shared_kv_tile (&k_smem)[PIPE_STAGES] = al.allocate<shared_kv_tile, PIPE_STAGES>();
    shared_kv_tile (&v_smem)[PIPE_STAGES] = al.allocate<shared_kv_tile, PIPE_STAGES>();
    shared_qo_tile (&qo_smem)[1] = al.allocate<shared_qo_tile, 1>();

    kv_tile<bf16> k_reg;
    qo_tile<bf16> q_reg;
    kv_tile<bf16, col_l> v_reg;
    qo_tile<float> o_reg;
    attn_tile<float> att_block;
    attn_tile<bf16> att_block_mma;
    typename attn_tile<float>::col_vec max_vec_last, max_vec, norm_vec;
    // going through shared memory improves coalescing of dram reads.
    load<1, false>(qo_smem[0], g.Qg, {batch, 0, head, 0});
    __syncwarp();
    load(q_reg, qo_smem[0]);
    __syncthreads();
    if constexpr(TILE_SIZE_D == 128) q_reg *= __float2bfloat16(0.08838834764f * 1.44269504089f);

    max_vec = base_types::constants<float>::neg_infty();
    norm_vec = 0.f;
    o_reg = 0.f;
    // launch the load of the first k, v tiles
    int kv_blocks = g.Kg.depth() / TILE_SIZE_N, tic = 0;
    load_async<1, false>(k_smem[0], g.Kg, {batch, 0, head, 0});
    load_async<1, false>(v_smem[0], g.Vg, {batch, 0, head, 0});
    for (auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic=(tic + 1) % PIPE_STAGES) {
        int next_load_idx = kv_idx + 1;
        if (next_load_idx * TILE_SIZE_N < g.Kg.depth()) {  // Remove the redundant multiplication with TILE_SIZE_N
            int next_tic = (tic + 1) % PIPE_STAGES;
            load_async<1, false>(k_smem[next_tic], g.Kg, {batch, next_load_idx, head, 0});
            load_async<1, false>(v_smem[next_tic], g.Vg, {batch, next_load_idx, head, 0});
            load_async_wait<1>();
        }
        else load_async_wait();
        __syncthreads();

        load(k_reg, k_smem[tic]);
        att_block = 0.f;
        mma<transpose::N, transpose::T>(att_block, q_reg, k_reg, att_block); // Q@K.T
        max_vec_last = max_vec;
        max_vec = max<axis::COL>(att_block, max_vec); 
        att_block = exp2(att_block - max_vec); 
        max_vec_last = exp2(max_vec_last - max_vec);
        norm_vec *= max_vec_last; 
        norm_vec = sum<axis::COL>(att_block, norm_vec); 
        att_block_mma = att_block; 
        load(v_reg, v_smem[tic]); 
        o_reg *= max_vec_last; 
        mma<transpose::N, transpose::N>(o_reg, att_block_mma, v_reg, o_reg);
    }

    o_reg /= norm_vec;
    __syncthreads();
    store(qo_smem[0], o_reg);
    __syncwarp();
    store<1, false>(g.Og, qo_smem[0], {batch, 0, head, 0});
}

void run_attend_ker(globals g) {
    unsigned long mem_size = (kittens::MAX_SHARED_MEMORY) / 2;// PIPE_STAGES * TILE_SIZE_N * TILE_SIZE_D * 2 * 2 + TILE_SIZE_M * TILE_SIZE_D * 2 * 2; 
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        attend_ker),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    hipDeviceSynchronize();
    attend_ker<<<dim3(g.Qg.rows(), g.Qg.batch()), WARP_THREADS, mem_size>>>(g);
    hipDeviceSynchronize();
    CudaCheckError();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return;
    }
}

PYBIND11_MODULE(test_04_15_09_58, m) {
    m.doc() = "test_04_15_09_58 python module";
    py::bind_function<run_attend_ker>(m, "wrapped_attend_ker", &globals::Qg, &globals::Kg, &globals::Vg, &globals::Og);
}