#include "hip/hip_runtime.h"
#include <iostream>
#include "kittens.cuh"
#include "pyutils/pyutils.cuh"

#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError( const char *file, const int line ) {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
}

using namespace kittens;

constexpr int PIPE_STAGES = 2;
constexpr int TILE_SIZE_N = 16;
// TILE_SIZE_M and TILE_SIZE_D are now template parameters

template<int M, int D>
struct attend_params {
    static_assert(M == 16 || M == 32, "TILE_SIZE_M must be either 16 or 32");
    static_assert(D == 64 || D == 128, "TILE_SIZE_D must be either 64 or 128");
    
    template<typename T=bf16, typename L=row_l> using kv_tile = rt<T, TILE_SIZE_N, D, L>;
    template<typename T=bf16, typename L=row_l> using qo_tile = rt<T, M, D, L>;
    template<typename T=float> using attn_tile = rt<T, M, TILE_SIZE_N>;
    using shared_kv_tile = st_bf<TILE_SIZE_N, D>;
    using shared_qo_tile = st_bf<M, D>;
    using global_qkvo_layout = gl<bf16, -1, -1, -1, D>; // batch, depth, row, col
};

template<int M, int D>
struct globals {
    typename attend_params<M, D>::global_qkvo_layout Qg, Kg, Vg, Og;
};

template<int M, int D>
__launch_bounds__(WARP_THREADS, 1)
__global__ void attend_ker(const __grid_constant__ globals<M, D> g) {
    using params = attend_params<M, D>;
    
    const int batch = blockIdx.y, head = blockIdx.x;

    extern __shared__ alignment_dummy __shm[];
    shared_allocator al((int*)&__shm[0]);

    typename params::shared_kv_tile (&k_smem)[PIPE_STAGES] = al.allocate<typename params::shared_kv_tile, PIPE_STAGES>();
    typename params::shared_kv_tile (&v_smem)[PIPE_STAGES] = al.allocate<typename params::shared_kv_tile, PIPE_STAGES>();
    typename params::shared_qo_tile (&qo_smem)[1] = al.allocate<typename params::shared_qo_tile, 1>();

    typename params::template kv_tile<bf16> k_reg;
    typename params::template qo_tile<bf16> q_reg;
    typename params::template kv_tile<bf16, col_l> v_reg;
    typename params::template qo_tile<float> o_reg;
    typename params::template attn_tile<float> att_block;
    typename params::template attn_tile<bf16> att_block_mma;
    typename params::template attn_tile<float>::col_vec max_vec_last, max_vec, norm_vec;
    
    // going through shared memory improves coalescing of dram reads.
    load<1, false>(qo_smem[0], g.Qg, {batch, 0, head, 0});
    __syncwarp();
    load(q_reg, qo_smem[0]);
    __syncthreads();
    if constexpr(D == 128) q_reg *= __float2bfloat16(0.08838834764f * 1.44269504089f);
    else if constexpr(D == 64) q_reg *= __float2bfloat16(0.125f * 1.44269504089f);

    max_vec = base_types::constants<float>::neg_infty();
    norm_vec = 0.f;
    o_reg = 0.f;
    // launch the load of the first k, v tiles
    int kv_blocks = g.Kg.depth() / TILE_SIZE_N, tic = 0;
    load_async<1, false>(k_smem[0], g.Kg, {batch, 0, head, 0});
    load_async<1, false>(v_smem[0], g.Vg, {batch, 0, head, 0});
    for (auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic=(tic + 1) % PIPE_STAGES) {
        int next_load_idx = kv_idx + 1;
        if (next_load_idx * TILE_SIZE_N < g.Kg.depth()) {  // Remove the redundant multiplication with TILE_SIZE_N
            int next_tic = (tic + 1) % PIPE_STAGES;
            load_async<1, false>(k_smem[next_tic], g.Kg, {batch, next_load_idx, head, 0});
            load_async<1, false>(v_smem[next_tic], g.Vg, {batch, next_load_idx, head, 0});
            load_async_wait<1>();
        }
        else load_async_wait();
        __syncthreads();

        load(k_reg, k_smem[tic]);
        att_block = 0.f;
        mma<transpose::N, transpose::T>(att_block, q_reg, k_reg, att_block); // Q@K.T
        max_vec_last = max_vec;
        max_vec = max<axis::COL>(att_block, max_vec); 
        att_block = exp2(att_block - max_vec); 
        max_vec_last = exp2(max_vec_last - max_vec);
        norm_vec *= max_vec_last; 
        norm_vec = sum<axis::COL>(att_block, norm_vec); 
        att_block_mma = att_block; 
        load(v_reg, v_smem[tic]); 
        o_reg *= max_vec_last; 
        mma<transpose::N, transpose::N>(o_reg, att_block_mma, v_reg, o_reg);
    }

    o_reg /= norm_vec;
    __syncthreads();
    store(qo_smem[0], o_reg);
    __syncwarp();
    store<1, false>(g.Og, qo_smem[0], {batch, 0, head, 0});
}

template<int M, int D>
void run_attend_ker(globals<M, D> g) {
    unsigned long mem_size = (kittens::MAX_SHARED_MEMORY) / 2;
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        attend_ker<M), D>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    hipDeviceSynchronize();
    attend_ker<M, D><<<dim3(g.Qg.rows(), g.Qg.batch()), WARP_THREADS, mem_size>>>(g);
    hipDeviceSynchronize();
    CudaCheckError();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return;
    }
}

PYBIND11_MODULE(test_04_15_09_58, m) {
    m.doc() = "test_04_15_09_58 python module";
    
    // Expose the different template specializations
    py::bind_function<run_attend_ker<16, 64>>(m, "wrapped_attend_ker_16_64", 
        &globals<16, 64>::Qg, &globals<16, 64>::Kg, &globals<16, 64>::Vg, &globals<16, 64>::Og);
    
    py::bind_function<run_attend_ker<16, 128>>(m, "wrapped_attend_ker_16_128", 
        &globals<16, 128>::Qg, &globals<16, 128>::Kg, &globals<16, 128>::Vg, &globals<16, 128>::Og);
    
    py::bind_function<run_attend_ker<32, 64>>(m, "wrapped_attend_ker_32_64", 
        &globals<32, 64>::Qg, &globals<32, 64>::Kg, &globals<32, 64>::Vg, &globals<32, 64>::Og);
    
    py::bind_function<run_attend_ker<32, 128>>(m, "wrapped_attend_ker_32_128", 
        &globals<32, 128>::Qg, &globals<32, 128>::Kg, &globals<32, 128>::Vg, &globals<32, 128>::Og);
}