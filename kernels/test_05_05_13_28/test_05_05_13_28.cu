#include "hip/hip_runtime.h"
#include <iostream>
#include "kittens.cuh"
#include "pyutils/pyutils.cuh"

#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError( const char *file, const int line ) {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
}

using namespace kittens;

constexpr int PIPE_STAGES = 2;
constexpr int TILE_SIZE_N = 16;
// TILE_SIZE_M and TILE_SIZE_D are now template parameters

template<int M, int D, int NUM_WORKERS>
struct attend_params {
    static_assert(M == 16 || M == 32 || M == 48 || M == 64, "TILE_SIZE_M must be either {16, 32, 48, 64}");
    static_assert(D == 64 || D == 96 || D == 128 || D == 160, "TILE_SIZE_D must be either {64, 96, 128, 160}");
    
    template<typename T=bf16, typename L=row_l> using kv_tile = rt<T, TILE_SIZE_N, D, L>;
    template<typename T=bf16, typename L=row_l> using qo_tile = rt<T, M, D, L>;
    template<typename T=float> using attn_tile = rt<T, M, TILE_SIZE_N>;
    using shared_kv_tile = st_bf<TILE_SIZE_N, D>;
    using shared_qo_tile = st_bf<M, D>;
    using global_qkvo_layout = gl<bf16, -1, -1, NUM_WORKERS, D>; // batch, depth, row, col
};

template<int M, int D, int NUM_WORKERS>
struct globals {
    typename attend_params<M, D, NUM_WORKERS>::global_qkvo_layout Qg, Kg, Vg, Og;
};

template<int M, int D, int NUM_WORKERS>
__launch_bounds__(NUM_WORKERS*WARP_THREADS, 1)
__global__ void attend_ker(const __grid_constant__ globals<M, D, NUM_WORKERS> g) {
    using params = attend_params<M, D, NUM_WORKERS>;
    
    const int batch = blockIdx.x, head = kittens::warpid();

    extern __shared__ alignment_dummy __shm[];
    shared_allocator al((int*)&__shm[0]);

    typename params::shared_kv_tile (&k_smem)[NUM_WORKERS][PIPE_STAGES] = al.allocate<typename params::shared_kv_tile, NUM_WORKERS, PIPE_STAGES>();
    typename params::shared_kv_tile (&v_smem)[NUM_WORKERS][PIPE_STAGES] = al.allocate<typename params::shared_kv_tile, NUM_WORKERS, PIPE_STAGES>();
    typename params::shared_qo_tile (&qo_smem)[NUM_WORKERS] = al.allocate<typename params::shared_qo_tile, NUM_WORKERS>();

    typename params::template kv_tile<bf16> k_reg;
    typename params::template qo_tile<bf16> q_reg;
    typename params::template kv_tile<bf16, col_l> v_reg;
    typename params::template qo_tile<float> o_reg;
    typename params::template attn_tile<float> att_block;
    typename params::template attn_tile<bf16> att_block_mma;
    typename params::template attn_tile<float>::col_vec max_vec_last, max_vec, norm_vec;
    
    // going through shared memory improves coalescing of dram reads.
    load<1, false>(qo_smem[head], g.Qg, {batch, 0, head, 0});
    __syncwarp();
    load(q_reg, qo_smem[head]);
    __syncthreads();
    if constexpr(D == 128) q_reg *= __float2bfloat16(0.08838834764f * 1.44269504089f);
    else if constexpr(D == 64) q_reg *= __float2bfloat16(0.125f * 1.44269504089f);
    else if constexpr(D == 96) q_reg *= __float2bfloat16(0.10206207262f * 1.44269504089f);
    else if constexpr(D == 160) q_reg *= __float2bfloat16(0.07905694151f * 1.44269504089f);

    max_vec = base_types::constants<float>::neg_infty();
    norm_vec = 0.f;
    o_reg = 0.f;
    // launch the load of the first k, v tiles
    int kv_blocks = g.Kg.depth() / TILE_SIZE_N, tic = 0;
    load_async<1, false>(k_smem[head][0], g.Kg, {batch, 0, head, 0});
    load_async<1, false>(v_smem[head][0], g.Vg, {batch, 0, head, 0});
    for (auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic=(tic + 1) % PIPE_STAGES) {
        int next_load_idx = kv_idx + 1;
        if (next_load_idx * TILE_SIZE_N < g.Kg.depth()) {  // Remove the redundant multiplication with TILE_SIZE_N
            int next_tic = (tic + 1) % PIPE_STAGES;
            load_async<1, false>(k_smem[head][next_tic], g.Kg, {batch, next_load_idx, head, 0});
            load_async<1, false>(v_smem[head][next_tic], g.Vg, {batch, next_load_idx, head, 0});
            load_async_wait<1>();
        }
        else load_async_wait();
        __syncthreads();

        load(k_reg, k_smem[head][tic]);
        att_block = 0.f;
        mma<transpose::N, transpose::T>(att_block, q_reg, k_reg, att_block); // Q@K.T
        max_vec_last = max_vec;
        max_vec = max<axis::COL>(att_block, max_vec); 
        att_block = exp2(att_block - max_vec); // M * Tn
        max_vec_last = exp2(max_vec_last - max_vec); // M
        norm_vec *= max_vec_last; 
        norm_vec = sum<axis::COL>(att_block, norm_vec);  // M * Tn
        att_block_mma = att_block; 
        load(v_reg, v_smem[tic]); 
        o_reg *= max_vec_last;  // M * d
        mma<transpose::N, transpose::N>(o_reg, att_block_mma, v_reg, o_reg);
    }

    o_reg /= norm_vec;
    __syncthreads();
    store(qo_smem[head], o_reg);
    __syncwarp();
    store<1, false>(g.Og, qo_smem[head], {batch, 0, head, 0});
}

template<int M, int D, int NUM_WORKERS>
void run_attend_ker(globals<M, D, NUM_WORKERS> g) {
    unsigned long mem_planned = 
    (2 * PIPE_STAGES * attend_params<M, D, NUM_WORKERS>::shared_kv_tile::num_elements * 
     sizeof(typename attend_params<M, D, NUM_WORKERS>::shared_kv_tile::dtype)) +
    (attend_params<M, D, NUM_WORKERS>::shared_qo_tile::num_elements * 
     sizeof(typename attend_params<M, D, NUM_WORKERS>::shared_qo_tile::dtype));
    
    unsigned long mem_size = mem_planned * NUM_WORKERS;
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        attend_ker<M), D, NUM_WORKERS>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    hipDeviceSynchronize();
    attend_ker<M, D, NUM_WORKERS><<<dim3(g.Qg.batch()), NUM_WORKERS * WARP_THREADS, mem_size>>>(g);
    hipDeviceSynchronize();
    CudaCheckError();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return;
    }
}

PYBIND11_MODULE(test_05_05_13_28, m) {
    m.doc() = "test_05_05_13_28 python module";
    
    // Expose the different template specializations
    py::bind_function<run_attend_ker<16, 64, 12>>(m, "wrapped_attend_ker_16_64_12", 
        &globals<16, 64, 12>::Qg, &globals<16, 64, 12>::Kg, &globals<16, 64, 12>::Vg, &globals<16, 64, 12>::Og);
    py::bind_function<run_attend_ker<16, 96, 8>>(m, "wrapped_attend_ker_16_96_8", 
        &globals<16, 96, 8>::Qg, &globals<16, 96, 8>::Kg, &globals<16, 96, 8>::Vg, &globals<16, 96, 8>::Og);  
    py::bind_function<run_attend_ker<16, 128, 7>>(m, "wrapped_attend_ker_16_128_7", 
        &globals<16, 128, 7>::Qg, &globals<16, 128, 7>::Kg, &globals<16, 128, 7>::Vg, &globals<16, 128, 7>::Og);
    py::bind_function<run_attend_ker<16, 160, 4>>(m, "wrapped_attend_ker_16_160_4", 
        &globals<16, 160, 4>::Qg, &globals<16, 160, 4>::Kg, &globals<16, 160, 4>::Vg, &globals<16, 160, 4>::Og);

    py::bind_function<run_attend_ker<32, 64, 8>>(m, "wrapped_attend_ker_32_64_8", 
        &globals<32, 64, 8>::Qg, &globals<32, 64, 8>::Kg, &globals<32, 64, 8>::Vg, &globals<32, 64, 8>::Og);
    py::bind_function<run_attend_ker<32, 96, 8>>(m, "wrapped_attend_ker_32_96_8", 
        &globals<32, 96, 8>::Qg, &globals<32, 96, 8>::Kg, &globals<32, 96, 8>::Vg, &globals<32, 96, 8>::Og);
    py::bind_function<run_attend_ker<32, 128, 6>>(m, "wrapped_attend_ker_32_128_6", 
        &globals<32, 128, 6>::Qg, &globals<32, 128, 6>::Kg, &globals<32, 128, 6>::Vg, &globals<32, 128, 6>::Og);
    py::bind_function<run_attend_ker<32, 160, 5>>(m, "wrapped_attend_ker_32_160_5", 
        &globals<32, 160, 5>::Qg, &globals<32, 160, 5>::Kg, &globals<32, 160, 5>::Vg, &globals<32, 160, 5>::Og);

    py::bind_function<run_attend_ker<48, 64, 8>>(m, "wrapped_attend_ker_48_64_8", 
        &globals<48, 64, 8>::Qg, &globals<48, 64, 8>::Kg, &globals<48, 64, 8>::Vg, &globals<48, 64, 8>::Og);
    py::bind_function<run_attend_ker<48, 96, 7>>(m, "wrapped_attend_ker_48_96_7", 
        &globals<48, 96, 7>::Qg, &globals<48, 96, 7>::Kg, &globals<48, 96, 7>::Vg, &globals<48, 96, 7>::Og);
    py::bind_function<run_attend_ker<48, 128, 5>>(m, "wrapped_attend_ker_48_128_5", 
        &globals<48, 128, 5>::Qg, &globals<48, 128, 5>::Kg, &globals<48, 128, 5>::Vg, &globals<48, 128, 5>::Og);
    py::bind_function<run_attend_ker<48, 160, 4>>(m, "wrapped_attend_ker_48_160_4", 
        &globals<48, 160, 4>::Qg, &globals<48, 160, 4>::Kg, &globals<48, 160, 4>::Vg, &globals<48, 160, 4>::Og);

    py::bind_function<run_attend_ker<64, 64, 8>>(m, "wrapped_attend_ker_64_64_8", 
        &globals<64, 64, 8>::Qg, &globals<64, 64, 8>::Kg, &globals<64, 64, 8>::Vg, &globals<64, 64, 8>::Og);
    py::bind_function<run_attend_ker<64, 96, 6>>(m, "wrapped_attend_ker_64_96_6", 
        &globals<64, 96, 6>::Qg, &globals<64, 96, 6>::Kg, &globals<64, 96, 6>::Vg, &globals<64, 96, 6>::Og);
    py::bind_function<run_attend_ker<64, 128, 4>>(m, "wrapped_attend_ker_64_128_4", 
        &globals<64, 128, 4>::Qg, &globals<64, 128, 4>::Kg, &globals<64, 128, 4>::Vg, &globals<64, 128, 4>::Og);
    py::bind_function<run_attend_ker<64, 160, 3>>(m, "wrapped_attend_ker_64_160_3", 
        &globals<64, 160, 3>::Qg, &globals<64, 160, 3>::Kg, &globals<64, 160, 3>::Vg, &globals<64, 160, 3>::Og);
}